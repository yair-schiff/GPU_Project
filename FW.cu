/*
 * Author: Yair Schiff
 * Project: Fall 2018 CSCI.GA 3033-004: GPUs
 * Instructor: Prof. Zahran
 *
 * Project Description: This project explores the efficient implementation of the Floyd-Warshall (FW) algorithm, a
 * solution for the All-Pairs-Shortest-Path (APSP) and Transitive Closure problems. This project will compare sequential
 * (CPU) and parallel (GPU) versions of the algorithm.
 */

#include <ctype.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

/*****************************************************************
 * Macros
*****************************************************************/
#define MAX_GRAPH 397020 // max graph size
#define MAX_BUF 1000 // integer size of buffer for file reading
#define index(i, j, N)  ((i)*(N)) + (j) // To index element (i,j) of a 2D array stored as 1D
// Macro for error checking cuda API calls
#define CUDA_ERROR_CHECK(err) {\
    if (err != hipSuccess) {\
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(1);\
    }\
}

/*****************************************************************
 * Forward declarations
*****************************************************************/
unsigned int convert(char *st);
void read_input(const char *fn, int *adj_matrix, unsigned int N);
void preprocess_graph(int *adj_matrix, int *go_to, unsigned int N);
void print_adj(int *adj_matrix, unsigned int N);
void print_path(int *adj_matrix, int *go_to, unsigned int N);
void print_path_recursive(int *go_to, unsigned int i, unsigned int j, unsigned int N);
void FW_sequential(int *adj_matrix, int *go_to, unsigned int N);
void FW_parallel(int *adj_matrix, int *go_to, unsigned int N);
/*****************************************************************/

/*****************************************************************
 * main method
*****************************************************************/
int main(int argc, char *argv[]) {
    // Check that correct number of command line arguments given
    if (argc != 5) {
        fprintf(stderr, "usage: FW_seq <input> <N>\n");
        fprintf(stderr, "input = file containing adjacency matrix for the graph\n");
        fprintf(stderr, "N = number for vertices from input graph to use\n");
        fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
        fprintf(stderr, "verbose = false: if flag is set (i.e. 1 is passed) then original adjacency matrix and APSP "
                        "solution will be printed.\n");
        exit(1);
    }

    // Parse command line arguments
    const char *input_file_name = argv[1]; // input file
    unsigned int N;  // Number of vertices to use
    N = convert(argv[2]);
    if (N > MAX_GRAPH) {
        fprintf(stderr, "Max graph size allowed %u x %u. Defaulting to this size.", MAX_GRAPH, MAX_GRAPH);
        N = MAX_GRAPH;
    }
    int type_of_device = 0; // CPU or GPU
    type_of_device = atoi(argv[3]);
    int c;
    int verbose = 0;
    verbose = atoi(argv[4]);

    // Allocate memory for NxN adjacency matrix
    int *adj_matrix;
    adj_matrix = (int *) calloc( N * N, sizeof(int));
    if (adj_matrix == NULL) {
        fprintf(stderr, "malloc for adjacency matrix of size %u x %u failed.", N, N);
        exit(1);
    }

    // Allocate memory for NxN go_to matrix:
    int *go_to;
    go_to = (int *) malloc(sizeof(int) * N * N);
    if (go_to == NULL) {
        fprintf(stderr, "malloc for go_to matrix of size %u x %u failed.", N, N);
        exit(1);
    }

    // Read input and populate edges
    read_input(input_file_name, adj_matrix, N);

    // Pre-process adjacency matrix and next index matrix
    preprocess_graph(adj_matrix, go_to, N);
    if (verbose) print_adj(adj_matrix, N);

    // Declare variables for tracking time
    double time_taken;
    clock_t clock_start, clock_end;

    // Dispatch FW to either sequential or parallel version based on flag passed in
    if (!type_of_device) { // The CPU sequential version
        clock_start = clock();
        FW_sequential(adj_matrix, go_to, N);
        clock_end = clock();
        time_taken = ((double) clock_end - clock_start) / CLOCKS_PER_SEC;
        printf("Time taken to run FW algorithm sequentially: %lf seconds\n", time_taken);
    }
    else { // The GPU version
        clock_start = clock();
        FW_parallel(adj_matrix, go_to, N);
        clock_end = clock();
        time_taken = ((double) clock_end - clock_start) / CLOCKS_PER_SEC;
        printf("Time taken to run FW algorithm in parallel: %lf seconds\n", time_taken);
    }

    // Print solution path between every pair of vertices
    if (verbose) print_path(adj_matrix, go_to, N);

    free(adj_matrix);
    free(go_to);
    return 0;
}

/*******************************************************************************************************************
 * Floyd-Warshall algorithm to solve APSP problem sequentially
 *******************************************************************************************************************/
void FW_sequential(int *adj_matrix, int *go_to, unsigned int N) {
    unsigned int i, j, k;
    for (k = 0; k < N; k++) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (adj_matrix[index(i, j, N)] > (adj_matrix[index(i, k, N)] + adj_matrix[index(k, j, N)])) {
                    adj_matrix[index(i, j, N)] = adj_matrix[index(i, k, N)] + adj_matrix[index(k, j, N)];
                    go_to[index(i, j, N)] = (int) k;
                }
            }
        }
    }
}

/*******************************************************************************************************************
 * Kernel for running inner double for-loops of FW in parallel
 *******************************************************************************************************************/
__global__
void FW_kernel(int *adj_matrix, int *go_to, unsigned int N, int k) {
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) { // Boundary check
        if (adj_matrix[index(i, j, N)] > (adj_matrix[index(i, k, N)] + adj_matrix[index(k, j, N)])) {
            adj_matrix[index(i, j, N)] = adj_matrix[index(i, k, N)] + adj_matrix[index(k, j, N)];
            go_to[index(i, j, N)] = k;
        }
    }
}

/*******************************************************************************************************************
 * Floyd-Warshall algorithm to solve APSP problem on GPU
 *******************************************************************************************************************/
void FW_parallel(int *adj_matrix, int *go_to, unsigned int N) {
    // Allocate memory on GPU for NxN adjacency and next index matrices
    int num_bytes = sizeof(int) * N * N;
    int *adj_matrix_d;
    int *go_to_d;
    hipError_t err = hipMalloc((void **) &adj_matrix_d, num_bytes);
    CUDA_ERROR_CHECK(err);
    err = hipMemcpy(adj_matrix_d, adj_matrix, num_bytes, hipMemcpyHostToDevice);
    CUDA_ERROR_CHECK(err);
    err = hipMalloc((void **) &go_to_d, num_bytes);
    CUDA_ERROR_CHECK(err);
    err = hipMemcpy(go_to_d, go_to, num_bytes, hipMemcpyHostToDevice);
    CUDA_ERROR_CHECK(err);

    // Get warp size from device properties and set it as block size
    hipDeviceProp_t dev_prop;
    err = hipGetDeviceProperties(&dev_prop, 0);
    CUDA_ERROR_CHECK(err);
    int warp_size = dev_prop.warpSize;
    int dim_helper = ceil(N/((double) warp_size));
    dim3 dimGrid(dim_helper, dim_helper);
    dim3 dimBlock(warp_size, warp_size);

    // Run FW triple-loop by launching a new kernel for each k
    unsigned int k;
    for (k = 0; k < N; k++) {
        FW_kernel<<<dimGrid, dimBlock>>>(adj_matrix_d, go_to_d, N, (int) k);
        err = hipGetLastError();
        CUDA_ERROR_CHECK(err);
    }

    // Copy solution back to host
    err = hipMemcpy(adj_matrix, adj_matrix_d, num_bytes, hipMemcpyDeviceToHost);
    CUDA_ERROR_CHECK(err);
    err = hipFree(adj_matrix_d);
    CUDA_ERROR_CHECK(err);
    err = hipMemcpy(go_to, go_to_d, num_bytes, hipMemcpyDeviceToHost);
    CUDA_ERROR_CHECK(err);
    err = hipFree(go_to_d);
    CUDA_ERROR_CHECK(err);
}

/*******************************************************************************************************************
 * Read input graph file and populate adjacency matrix
 *******************************************************************************************************************/
void read_input(const char *fn, int *adj_matrix, unsigned int N) {
    const char *fileName = fn;
    FILE *input = fopen(fileName, "r");
    if (input == NULL) {
        fprintf(stderr, "Error while opening the file.\n");
        exit(1);
    }

    char buffer[MAX_BUF];

    // Read file
    int line = 0;
    while (1) {
        line++;
        fgets(buffer, MAX_BUF, input); // get next line
        // Skip lines starting with '#' and empty lines
        if (buffer[0] == '#' || buffer[0] == '\n' || buffer[0] == ' ') continue;
        int i; // row
        int j; // column
        int rel; // relationship (take absolute value, below)
        int rc = sscanf(buffer, "%d|%d|%d",&i, &j, &rel);
        if (rc != 3) {
            fprintf(stderr, "Input file not well formatted (Line %d). "
                            "Expected format of graph lines: <v1>|<v2>|<edge>.\n", line);
            exit(1);
        }
        if (i <= N && j <= N) adj_matrix[index(i-1, j-1, N)] = abs(rel);
        if (feof(input)) break;
    }

    // Close file
    fclose(input);
}

/*******************************************************************************************************************
 * Convert command line input to integer
 * Code taken from https://stackoverflow.com/questions/34206446/how-to-convert-string-into-unsigned-int-c
 *******************************************************************************************************************/
unsigned int convert(char *st) {
    char *x;
    for (x = st ; *x ; x++) {
        if (!isdigit(*x))
            return 0L;
    }
    return (strtoul(st, 0L, 10));
}

/*******************************************************************************************************************
 * Pre-process adjacency matrix and next index matrix:
 * Fill non-edges with int_max/2 in adjacency matrix and -1 in next index on path matrix
 *******************************************************************************************************************/
void preprocess_graph(int *adj_matrix, int *go_to, unsigned int N) {
    unsigned int i, j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            if (adj_matrix[index(i, j, N)] >= 1) {
                go_to[index(i, j, N)] = j;
            }
            else {
                adj_matrix[index(i, j, N)] = INT_MAX / 2;
                go_to[index(i, j, N)] = -1;
            }
        }
    }
}

/*******************************************************************************************************************
 * Print adjacency matrix read in from file
 *******************************************************************************************************************/
void print_adj(int *adj_matrix, unsigned int N) {
    unsigned int i, j;
    printf("Original adjacency matrix:\n");
    printf("    |");
    for (i = 0; i < N; i++) printf(" %2d |", i+1);
    printf("\n----|----|----|----|----|----|----|----|----|----|----|\n");
    for (i = 0; i < N; i++) {
        printf(" %2d |", i+1);
        for (j = 0; j < N; j++) {
            if (adj_matrix[index(i, j, N)] != INT_MAX/2) printf(" %2d |", adj_matrix[index(i, j, N)]);
            else printf("  - |");
        }
        printf("\n----|----|----|----|----|----|----|----|----|----|----|\n");
    }
}

/*******************************************************************************************************************
 * Print path between all vertex pairs i,j
 *******************************************************************************************************************/
void print_path(int *adj_matrix, int *go_to, unsigned int N) {
    unsigned int i, j;
    printf("\nAPSP solution:\n");
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            if (go_to[index(i, j, N)] == -1) {
                printf("No path exists between %u and %u.\n", i+1, j+1);
            }
            else {
                printf("Path from %u to %u (length: %d): %u", i+1, j+1, adj_matrix[index(i, j, N)], i+1);
                print_path_recursive(go_to, i, j, N);
                printf("\n");
            }
        }
    }
}

/*******************************************************************************************************************
 * Recursive method for printing path
 *******************************************************************************************************************/
void print_path_recursive(int *go_to, unsigned int i, unsigned int j, unsigned int N) {
    unsigned int next = go_to[index(i, j, N)];
    if (next == j) {
        printf("->%u", next+1);
        return;
    }
    else {
        print_path_recursive(go_to, i, next, N);
        print_path_recursive(go_to, next, j, N);
    }
}